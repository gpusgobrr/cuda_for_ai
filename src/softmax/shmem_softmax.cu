#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <torch/torch.h>
#include "shmem_softmax.cuh"


// CUDA kernel for softmax operation
__global__ void shmem_softmax_kernel(float *input, float *output, const int batch_size, const int num_elements) {
    extern __shared__ float shared_data[]; // Shared memory for intermediate results
    int row = blockIdx.x; // Each block processes one row
    int tid = threadIdx.x;
    // printf("-------- Row: %d, Thread: %d --------- \n", row, tid);
    if (row < batch_size) {
        // Load row into shared memory
        float max_val = -INFINITY;
        for (int i = tid; i < num_elements; i += 1) {
            shared_data[i] = input[row * num_elements + i];
            max_val = fmaxf(max_val, shared_data[i]);
            // printf("Row: %d, Thread: %d, Value: %f\n", row, tid, shared_data[i]);
        }
        __syncthreads();

        // printf("Row: %d, Thread: %d, Max Value: %f\n", row, tid, max_val);

        // Compute exponentials and sum
        float sum = 0.0f;
        for (int i = tid; i < num_elements; i += 1) {
            shared_data[i] = expf(shared_data[i] - max_val);
            sum += shared_data[i];
            // printf("Row: %d, Thread: %d, Max Value: %f, Exp Value: %f\n, Sum: %f\n", row, tid, max_val, shared_data[i], sum);
        }

        // printf("Row: %d, Thread: %d, Sum before sync: %f\n", row, tid, sum);
        __syncthreads();

        // Normalize and write back
        for (int i = tid; i < num_elements; i += 1) {
            output[row * num_elements + i] = shared_data[i] / sum;
            // printf("Row: %d, Thread: %d, Output Value: %f\n", row, tid, output[row * num_elements + i]);
        }
    }
}

// Host function to launch the kernel with PyTorch tensors
torch::Tensor shmem_softmax(const torch::Tensor &input)
{
    auto output = torch::empty_like(input);
    // Make sure tensors are on CUDA
    TORCH_CHECK(input.device().is_cuda(), "Input tensor must be on CUDA device");
    TORCH_CHECK(output.device().is_cuda(), "Output tensor must be on CUDA device");
    TORCH_CHECK(input.dtype() == torch::kFloat32, "Input tensor must be float32");
    TORCH_CHECK(output.dtype() == torch::kFloat32, "Output tensor must be float32");

    // Extract dimensions
    TORCH_CHECK(input.dim() == 2, "Input tensor must be 2D (batch_size × num_elements)");
    const int batch_size = static_cast<int>(input.size(0));
    const int sequence_length = static_cast<int>(input.size(1));

    // Get raw pointers to the tensor data
    const auto d_input = input.data_ptr<float>();
    const auto d_output = output.data_ptr<float>();

    size_t threads = std::min<size_t>(1024, 1 << static_cast<int>(std::floor(std::log2(sequence_length))));
    size_t blocks_per_grid = batch_size;
    // std::cout << "Threads: " << threads  << " | Blocks per grid: " << blocks_per_grid << std::endl;

    shmem_softmax_kernel<<<blocks_per_grid, threads, sequence_length * sizeof(float)>>>(
        d_input, d_output, batch_size, sequence_length
    );
    return output;
}
